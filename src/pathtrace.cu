#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

// sort by material
#define SORTMATERIAL 1

// Russian roulette
#define RUSSIAN_ROULETTE 1
#define MIN_BOUNCES 3
#define MIN_SURVIVAL_PROB 0.05f

// AABB Bounding test
#define USE_AABB 1

// BVH
#define USE_BVH 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

// Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

// implement SampleUniformDiskConcentric from PBRT
__host__ __device__ glm::vec2 sampleUniformDiskConcentric(glm::vec2 u) {
    // Map u to [-1,1]^2
    glm::vec2 uOffset = 2.0f * u - glm::vec2(1.0f);
    if (uOffset.x == 0 && uOffset.y == 0)
        return glm::vec2(0.0f);

    float theta, r;
    if (abs(uOffset.x) > abs(uOffset.y)) {
        r = uOffset.x;
        theta = (PI / 4.0f) * (uOffset.y / uOffset.x);
    } else {
        r = uOffset.y;
        theta = (PI / 2.0f) - (PI / 4.0f) * (uOffset.x / uOffset.y);
    }

    return r * glm::vec2(cos(theta), sin(theta));
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;

static glm::vec3* dev_vertices = NULL;
static glm::vec3* dev_normals = NULL;
static glm::vec2* dev_uvs = NULL;
static Triangle* dev_triangles = NULL;
static BVHNode* dev_BVHNodes = NULL;

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_vertices, scene->vertices.size() * sizeof(glm::vec3));
    hipMemcpy(dev_vertices, scene->vertices.data(), scene->vertices.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);

    hipMalloc(&dev_normals, scene->normals.size() * sizeof(glm::vec3));
    hipMemcpy(dev_normals, scene->normals.data(), scene->normals.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);

    hipMalloc(&dev_uvs, scene->uvs.size() * sizeof(glm::vec2));
    hipMemcpy(dev_uvs, scene->uvs.data(), scene->uvs.size() * sizeof(glm::vec2), hipMemcpyHostToDevice);

    hipMalloc(&dev_triangles, scene->triangles.size() * sizeof(Triangle));
    hipMemcpy(dev_triangles, scene->triangles.data(), scene->triangles.size() * sizeof(Triangle), hipMemcpyHostToDevice);

#if USE_BVH
    hipMalloc(&dev_BVHNodes, scene->bvh.size() * sizeof(BVHNode));
    hipMemcpy(dev_BVHNodes, scene->bvh.data(), scene->bvh.size() * sizeof(BVHNode), hipMemcpyHostToDevice);
#endif

    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);

    hipFree(dev_vertices);
    hipFree(dev_normals);
    hipFree(dev_uvs);
    hipFree(dev_triangles);

#if USE_BVH
    hipFree(dev_BVHNodes);
#endif

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
        segment.insideObject = false;

        // Stochastic Antialiasing
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
        thrust::uniform_real_distribution<float> u01(0, 1);
        float jitterX = u01(rng) - 0.5f;
        float jitterY = u01(rng) - 0.5f;

        glm::vec3 rayDirection = glm::normalize(cam.view
           - cam.right * cam.pixelLength.x * ((float)x + jitterX - (float)cam.resolution.x * 0.5f)
           - cam.up * cam.pixelLength.y * ((float)y + jitterY - (float)cam.resolution.y * 0.5f)
        );

        // Apply depth of field
        if (cam.aperture > 0.0f) {
            // Sample point on the lens aperture using concentric mapping
            glm::vec2 pLens = cam.aperture * sampleUniformDiskConcentric(glm::vec2(u01(rng), u01(rng)));
            glm::vec3 lensOffset = pLens.x * cam.right + pLens.y * cam.up;

            // Compute focal point
            float t = cam.focalDistance / glm::dot(rayDirection, glm::normalize(cam.view));
            glm::vec3 focalPoint = cam.position + rayDirection * t;

            // Adjust ray origin and direction
            segment.ray.origin = cam.position + lensOffset;
            segment.ray.direction = glm::normalize(focalPoint - segment.ray.origin);
        } else {
            segment.ray.direction = rayDirection;
        }
    }
}

// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    ShadeableIntersection* intersections,
    glm::vec3* vertices,
    glm::vec3* normals,
    glm::vec2* uvs,
    Triangle* triangles,
    BVHNode* BVHNodes)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        glm::vec2 uv;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;
        glm::vec2 tmp_uv;
        bool tmp_outside = true;

        // naive parse through global geoms
        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, tmp_outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, tmp_outside);
            } else if (geom.type == MESH)
            {
#if USE_BVH
                t = meshIntersectionTestWithBVH(geom,
                                            pathSegment.ray,
                                            tmp_intersect,
                                            tmp_normal,
                                            tmp_outside,
                                            vertices,
                                            normals,
                                            uvs,
                                            triangles,
                                            BVHNodes);
#else
                // AABB Bounding test
//                if (USE_AABB) {
//                    if (!intersectRayAABB(pathSegment.ray, geom.minBounds, geom.maxBounds)) {
//                        continue;
//                    }
//                }

                t = meshIntersectionTest(geom,
                                         pathSegment.ray,
                                         tmp_intersect,
                                         tmp_normal,
                                         tmp_outside,
                                         vertices,
                                         normals,
                                         uvs,
                                         triangles);
#endif
            }

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
                uv = tmp_uv;
                outside = tmp_outside;
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            // The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            intersections[path_index].surfaceNormal = normal;
            intersections[path_index].uv = uv;
            intersections[path_index].outside = outside;
            intersections[path_index].intersectionPoint = intersect_point;
        }
    }
}

__global__ void shadeMaterial(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) return;

    // Preload frequently accessed variables into registers
    PathSegment& segment = pathSegments[idx];
    int remainingBounces = segment.remainingBounces;
    if (remainingBounces <= 0) return;

    ShadeableIntersection intersection = shadeableIntersections[idx];

    if (intersection.t > 0.0f) { // If the intersection exists...
        // Set up RNG
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);

        Material material = materials[intersection.materialId];
        glm::vec3 materialColor = material.color;

        if (material.emittance > 0.0f) {
            segment.color *= (materialColor * material.emittance);
            segment.remainingBounces = 0;  // Terminate ray if it hits a light source
        } else {
            glm::vec3 intersectionPoint = intersection.intersectionPoint;
            segment.insideObject = !intersection.outside;

            scatterRay(segment, intersectionPoint, intersection.surfaceNormal, material, rng);

            segment.remainingBounces--;

           if (segment.remainingBounces == 0) {
               segment.color = glm::vec3(0.0f);
           }

            // Apply Russian roulette
            if (RUSSIAN_ROULETTE && segment.remainingBounces >= MIN_BOUNCES) {
                // Compute the maximum component of the path throughput (color)
                float maxComponent = fmaxf(segment.color.r, fmaxf(segment.color.g, segment.color.b));
                maxComponent = glm::clamp(maxComponent, 0.0f, 1.0f);

                // Calculate termination probability q
                float q = fmaxf(1.0f - maxComponent, MIN_SURVIVAL_PROB);

                thrust::uniform_real_distribution<float> u01(0.0f, 1.0f);
                float randVal = u01(rng);

                if (randVal < q) {
                    // Terminate the path
                    segment.color = glm::vec3(0.0f);
                    segment.remainingBounces = 0;
                } else {
                    // Survive
                    segment.color /= (1.0f - q);
                }
            }
        }
    } else {
        // If no intersection, black out the ray
        segment.color = glm::vec3(0.0f);
        segment.remainingBounces = 0;
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];

        // Clamp the color
        glm::vec3 color = iterationPath.color;
        float maxColorValue = 10.0f;
        color = glm::min(color, glm::vec3(maxColorValue));

        image[iterationPath.pixelIndex] += color;
    }
}

struct IsActive {
    __host__ __device__
    bool operator()(const PathSegment& path) {
        return path.remainingBounces > 0;
    }
};

struct CompareByMaterial {
    __host__ __device__
    bool operator()(const ShadeableIntersection& a, const ShadeableIntersection& b) const {
        return a.materialId < b.materialId;
    }
};

void sortByMaterial(int num_paths) {
    thrust::sort_by_key(
            thrust::device,
            dev_intersections,
            dev_intersections + num_paths,
            dev_paths,
            CompareByMaterial()
    );
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 256;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    while (!iterationComplete)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            hst_scene->geoms.size(),
            dev_intersections,
            dev_vertices,
            dev_normals,
            dev_uvs,
            dev_triangles,
            dev_BVHNodes
        );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

        // TODO:
        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // TODO: compare between directly shading the path segments and shading
        //  path segments that have been reshuffled to be contiguous in memory.

        if (SORTMATERIAL) {
            sortByMaterial(num_paths);
        }

        shadeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials);

        // Stream compaction
        dev_path_end = thrust::partition(thrust::device, dev_paths, dev_path_end, IsActive());
        num_paths = dev_path_end - dev_paths;

        iterationComplete = depth > traceDepth || num_paths == 0;

        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
